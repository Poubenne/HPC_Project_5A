#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include "utility_functions.cuh"


/*
    * @brief Sorts small arrays with an exact number of blocks/threads
*/
__global__ void SortSmall_k(int **M, int j, int k) {
// Each block sorts one array

//  This is the thread's position in the array
    int tidx = threadIdx.x;

// This is the id of the table that the thread is going to help sorting
    int bid = blockIdx.x;

    //index of i + j
    int swap_index = tidx^j;

    // The threads with the lowest ids sort the array.
    if ((swap_index)>tidx) {
        if ((tidx&k)==0) {
            // Sort the array in ascending order
            if (M[bid][tidx]>M[bid][swap_index]) {
                // Swap index i and i+j
                int temp = M[bid][tidx];
                M[bid][tidx] = M[bid][swap_index];
                M[bid][swap_index] = temp;
            }
        }
        if ((tidx&k)!=0) {
            // Sort the array in descending order
            if (M[bid][tidx]<M[bid][swap_index]) {
                // Swap index i and i+j
                int temp = M[bid][tidx];
                M[bid][tidx] = M[bid][swap_index];
                M[bid][swap_index] = temp;
            }
        }
    }
}

/*
    * @brief Sorts small arrays
    * @param M The array of small arrays
    * @param N The number of small arrays
    * @param d The size of the small arrays (they all have the same)
    * @return A float corresponding to the time taken by the sort
*/
float SortSmall(int **M, size_t N, size_t d) {
//Sorts a group of small arrays using GPU parallelized Bitonic Sort

    int **M_GPU;
    hipMalloc(&M_GPU, N * sizeof(int*));
    int** tempo_array;
    tempo_array = (int**) malloc(N*sizeof(int*));
    for (int i = 0; i < N; i++){
        hipMalloc(&tempo_array[i], d * sizeof(int));
        hipMemcpy(tempo_array[i], M[i], d * sizeof(int), hipMemcpyHostToDevice);
    }    
    hipMemcpy(M_GPU, tempo_array, N*sizeof(int*), hipMemcpyHostToDevice);

//  Timing variables
    float elapsed_time;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int j, k;
    hipEventRecord(start, 0);
    // Progressively increase the size of major bitonic sorted array with each iteration
    for (k = 2; k <= d; k *= 2) {
        //Decrease the sorting step for each iteration to be able to correctly sort major size k arrays from current iteration
        for (j=k/2; j>0; j/=2) {
            SortSmall_k<<<N, d>>>(M_GPU, j, k);
        }
    }
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time, start, stop);


    for (size_t i=0;i<N;i++) {
        hipMemcpy(M[i], tempo_array[i], d * sizeof(int), hipMemcpyDeviceToHost);
    }

    for (int i = 0; i < N; i++){
        hipFree(tempo_array[i]);
    }
    free(tempo_array);
    hipFree(M_GPU);

    return elapsed_time;
}

int main(void) {
    size_t N = 1024;
    size_t d;
    size_t n_tries = 1000;
    int** array;
    
    array = (int**) malloc(N*sizeof(int*));
    
    float total_time;
    
    printf("Measuring the performance on different d values :\n");
    for (d=4; d<=1024; d*=2) {
        
        total_time = 0.0;
        for (size_t i=0;i<n_tries;i++) {
            for (size_t i=0;i<N;i++) {
                GenerateUnsortedRandomArray(array+i, d);
            }

            total_time += SortSmall(array, N, d);

            for (size_t i=0;i<N;i++) {
                if (! IsSortedAscending(array[i],d) ) {
                    printf("A sub-array isn't sorted... i = %zd\n", i);
                    exit(EXIT_FAILURE);
                }
            }

            for (size_t i=0;i<N;i++) {
                free(array[i]);
            }
        }
        printf("d : %zu, time for %zu runs : %2f s, average time : %2f s\n", d,  n_tries, total_time / 1000.0, (total_time / 1000.0) / n_tries);
    }
    printf("Done without errors!\n");



    free(array);


}