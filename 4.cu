#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include "utility_functions.cuh"


/*
    * @brief Sorts small arrays with an exact number of blocks/threads
*/
__global__ void SortSmall_k(int **M, int j, int k) {
// Each block sorts one array

//  This is the thread's position in the array
    int tidx = threadIdx.x;

// This is the id of the table that the thread is going to help sorting
    int bid = blockIdx.x;


// HERE DUNNO WHY THIS IS NOT NECESSARY
//    if (tidx >= NM[bid] || k >= NM[bid])
    //if (tidx >= NM[bid])
    //    return;


    int ixj = tidx^j;

    /* The threads with the lowest ids sort the array. */
    if ((ixj)>tidx) {
        if ((tidx&k)==0) {
        /* Sort ascending */
        if (M[bid][tidx]>M[bid][ixj]) {
            /* exchange(i,ixj); */
            int temp = M[bid][tidx];
            M[bid][tidx] = M[bid][ixj];
            M[bid][ixj] = temp;
        }
        }
        if ((tidx&k)!=0) {
        /* Sort descending */
        if (M[bid][tidx]<M[bid][ixj]) {
            /* exchange(i,ixj); */
            int temp = M[bid][tidx];
            M[bid][tidx] = M[bid][ixj];
            M[bid][ixj] = temp;
        }
        }
    }
}

/*
    * @brief Sorts small arrays
    * @param M The array of small arrays
    * @param N The number of small arrays
    * @param d The size of the small arrays (they all have the same)
*/
void SortSmall(int **M, size_t N, size_t d) {
//Sorts a group of small arrays using GPU parallelized Bitonic Sort

    int **M_GPU;
    hipMalloc(&M_GPU, N * sizeof(int*));
    int** tempo_array;
    tempo_array = (int**) malloc(N*sizeof(int*));
    for (int i = 0; i < N; i++){
        hipMalloc(&tempo_array[i], d * sizeof(int));
        hipMemcpy(tempo_array[i], M[i], d * sizeof(int), hipMemcpyHostToDevice);
    }
    
    hipMemcpy(M_GPU, tempo_array, N*sizeof(int*), hipMemcpyHostToDevice);

    int j, k;
    /* Major step */
    for (k = 2; k <= d; k <<= 1) {
        /* Minor step */
        for (j=k>>1; j>0; j=j>>1) {
            SortSmall_k<<<N, d>>>(M_GPU, j, k);
        }
    }

    for (size_t i=0;i<N;i++) {
        hipMemcpy(M[i], tempo_array[i], d * sizeof(int), hipMemcpyDeviceToHost);
    }

    for (int i = 0; i < N; i++){
        hipFree(tempo_array[i]);
    }
    hipFree(M_GPU);
}

int main(void) {
    size_t N = 1024;
    size_t d = 64;
    int** array;
    
    array = (int**) malloc(N*sizeof(int*));
    
    float TimerV;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    
    
    printf("Measuring the performance on different d values :\n\tWarning! the performance measure takes into account the memory allocation time\n");
    for (d=4; d<=1024; d*=2) {
        for (size_t i=0;i<N;i++) {
            GenerateUnsortedRandomArray(array+i, d);
        }

        hipEventRecord(start, 0);
        SortSmall(array, N, d);
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&TimerV, start, stop);
        printf("d : %zu, time : %2f s\n", d,TimerV / 1000.0);

        for (size_t i=0;i<N;i++) {
            if (! IsSortedAscending(array[i],d) ) {
                printf("A sub-array isn't sorrted... i = %zd\n", i);
                exit(EXIT_FAILURE);
            }
        }

        for (size_t i=0;i<N;i++) {
            free(array[i]);
        }
    }
    printf("Done without errors!\n");



    free(array);


}