#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <time.h>

size_t N=1024;
int d = 8;

void testCUDA(hipError_t error, const char *file, int line)  {
	if (error != hipSuccess) {
	   printf("There is an error in file %s at line %d\n", file, line);
       exit(EXIT_FAILURE);
	} 
}

#define testCUDA(error) (testCUDA(error, __FILE__ , __LINE__))

void QuickSort( int* array , size_t end) {
    if (end > 1) {
        int pivot = array[end-1];

        size_t cursor = 0;
        int swap_variable;
        for (size_t i = 0; i < end - 1 ; i++) {
            if ( array[i] < pivot ) {
                swap_variable = array[i];
                array[i] = array[cursor];
                array[cursor] = swap_variable;

                cursor = cursor + 1;
            }
        }
        swap_variable = array[end-1];
        array[end-1] = array[cursor];
        array[cursor] = swap_variable;

        QuickSort(array , cursor );
        QuickSort(array + cursor + 1, end - cursor - 1);
    }
}

bool IsSortedAscending(int* array, size_t length) {
    for(size_t i=1; i<length; i++) {
        if (array[i-1]>array[i]) {
            return false;
        }
    }
    return true;
}

bool IsSortedDescending(int* array, size_t length) {
    for(size_t i=1; i<length; i++) {
        if (array[i-1]<array[i]) {
            return false;
        }
    }
    return true;
}
/*
    * @brief Generates a random array
    * @param arr the array that'll be malloced and filled
    * @param The size of the array
    */
void GenerateRandomArray( int** arr, const size_t size ) {
    *arr = (int*) malloc(size*sizeof(int));

    for (size_t i=0 ; i<size ; i++) {
        (*arr)[i] = rand()%(5*size);
    }
    QuickSort(*arr, size);
}
void PrintList(int* A, size_t N){
    
    for (int i = 0; i < N; i++) {
        printf("%d ", A[i]);
    }
    printf("\n");
}



__global__ void mergeSmallBatch_k(const int** A, const int** B, int** M, const size_t* NA, const size_t* NB) {
//  Since the number of threads per block is a multiple of d, that means a single block can merge several arrays

//  This is the thread's position in the array
    int tidx = threadIdx.x%d;
//  This is the index of the array from the arrays given to a specific block
    int Qt = (threadIdx.x-tidx)/d;
//  This is the "global" index of the array, in which the thread is going to work
    int gbx = Qt + blockIdx.x*(blockDim.x/d);

//  the thread works on M[gbx][tidx]

    if ( gbx <= N ) {// excedent block
        return;
    }

/*    if (idx >= NA + NB)
        return;*/

    int K[2];
    int P[2];

    if (idx > NA[gbx]) {
        K[0] = tidx - NA[gbx];
        K[1] = NA[gbx];

        P[0] = NA[gbx];
        P[1] = tidx - NA[gbx];
    } else {
        K[0] = 0;
        K[1] = tidx;

        P[0] = tidx;
        P[1] = 0;
    }

    while (true) {
        int offset = abs((K[1] - P[1]) / 2);
        int Q[] = {K[0] + offset, K[1] - offset};

        if (Q[1] >= 0 && Q[0] <= NB[gbx] && (Q[1] == NA[gbx] || Q[0] == 0 || A[gbx][Q[1]] > B[Q[0] - 1])) {
            if (Q[0] == NB[gbx] || Q[1] == 0 || A[gbx][Q[1] - 1] <= B[gbx][Q[0]]) {
                if (Q[1] < NA[gbx] && (Q[0] == NB[gbx] || A[gbx][Q[1]] <= B[gbx][Q[0]]))
                    M[gbx][idx] = A[gbx][Q[1]];
                else
                    M[gbx][idx] = B[gbx][Q[0]];
                break;
            } else {
                K[0] = Q[0] + 1;
                K[1] = Q[1] - 1;
            }
        } else {
            P[0] = Q[0] - 1;
            P[1] = Q[1] + 1;
        }
    }
}



int main()
{
    srand(time(NULL));
    int a_sizes[] = {256, 512, 768};

    int** A = (int**) malloc(N*sizeof(int*));
    size_t* NA = (int*) malloc(N*sizeof(size_t));
    for (size_t i=0;i<N;i++) {
        NA[i] = a_sizes[i%3];
        A[i] = (int*) malloc(NA[i]*sizeof(int));
        GenerateRandomArray(&A, NA[i]);
    }


    int** B = (int**) malloc(N*sizeof(int*));
    size_t* NB = (int*) malloc(N*sizeof(size_t));
    for (size_t i=0;i<N;i++) {
        NB[i] = 1024 - NA[i];
        B[i] = (int*) malloc(NB[i]*sizeof(int));
        GenerateRandomArray(&B, NB[i]);
    }

    int** M = (int**) malloc(N * sizeof(int*));
    for (size_t i=0;i<N;i++) {
        M[i] = (int*) malloc((NB[i]+NA[i])*sizeof(size_t));
    }


//  We could create contiguous arrays...
    int** A_GPU;
    int** B_GPU;
    int** M_GPU;
    testCUDA(hipMalloc(&A_GPU, N * sizeof(int*)));
    testCUDA(hipMalloc(&B_GPU, N * sizeof(int*)));
    testCUDA(hipMalloc(&M_GPU, N * sizeof(int*)));


    for (size_t i=0;i<N;i++) {
        testCUDA(hipMalloc(M_GPU+i, 1024 * sizeof(int)));
        testCUDA(hipMalloc(A_GPU+i, NA[i] * sizeof(int)));
        testCUDA(hipMalloc(B_GPU+i, NB[i] * sizeof(int)));

        testCUDA(hipMemcpy(A_GPU[i], A[i], NA[i] * sizeof(int), hipMemcpyHostToDevice));
        testCUDA(hipMemcpy(B_GPU[i], B[i], NB[i] * sizeof(int), hipMemcpyHostToDevice));
    }





    int N_Blocks = N/d+10;
    int NTPB = 1024;
//    int NTPB = 1024;

    mergeSmallBatch_k<<<N_Blocks, NTPB>>>(A_GPU, B_GPU, M_GPU, NA, NB);

    testCUDA(hipMemcpy(M, M_GPU, (NA + NB) * sizeof(int), hipMemcpyDeviceToHost));

//    PrintList(M, (NA + NB));

    for (size_t i=0;i<N;i++) {
        if (! IsSortedAscending(M[i]) ) {
            printf("aïe aïe aïe...\n");
            exit(EXIT_FAILURE);
        }
    }



    testCUDA(hipFree(A_GPU));
    testCUDA(hipFree(B_GPU));
    testCUDA(hipFree(M_GPU));
    

    for (size_t i=0;i<N;i++) {
        free(A[i]);
        free(B[i]);
        free(M[i]);
    }
    free(A);
    free(B);
    free(M);

    return 0;
}